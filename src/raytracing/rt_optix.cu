
#include <hip/hip_runtime.h>
#ifdef GPROSHAN_OPTIX_FAIL


#include "mesh/che.h"
#include "mesh/vertex.cuh"
#include "raytracing/rt_optix_params.h"


#include <optix_device.h>
#include <cuda_runtime.h>


// geometry processing and shape analysis framework
namespace gproshan::rt {


extern "C" __constant__ launch_params params;

static __forceinline__ __device__
void * unpackPointer(uint32_t i0, uint32_t i1)
{
	const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
	void * ptr = reinterpret_cast<void*>(uptr);
	return ptr;
}

static __forceinline__ __device__
void packPointer(void * ptr, uint32_t & i0, uint32_t & i1)
{
	const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
	i0 = uptr >> 32;
	i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__ T * getPRD()
{
	const uint32_t u0 = optixGetPayload_0();
	const uint32_t u1 = optixGetPayload_1();
	return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

//------------------------------------------------------------------------------
// closest hit and anyhit programs for radiance-type rays.
//
// Note eventually we will have to create one pair of those for each
// ray type and each geometry type we want to render; but this
// simple example doesn't use any actual geometries yet, so we only
// create a single, dummy, set of them (we do have to have at least
// one group of them to set up the SBT)
//------------------------------------------------------------------------------

extern "C" __global__ void __closesthit__shadow()
{
	/* not going to be used ... */
}

extern "C" __global__ void __closesthit__radiance()
{
	const CHE & sbtData = *(const CHE *) optixGetSbtDataPointer();

	// ------------------------------------------------------------------
	// gather some basic hit information
	// ------------------------------------------------------------------
	const int primID = optixGetPrimitiveIndex();
	const index_t he = primID * che::mtrig;
	const float u = optixGetTriangleBarycentrics().x;
	const float v = optixGetTriangleBarycentrics().y;

	// ------------------------------------------------------------------
	// compute normal, using either shading normal (if avail), or
	// geometry normal (fallback)
	// ------------------------------------------------------------------
	const vertex_cu & A = sbtData.GT[sbtData.VT[he]];
	const vertex_cu & B = sbtData.GT[sbtData.VT[he + 1]];
	const vertex_cu & C = sbtData.GT[sbtData.VT[he + 2]];

	vertex_cu Ng = (B - A) * (C - A);
	vertex_cu Ns = (sbtData.VN)
		? ((1.f-u-v) * sbtData.VN[sbtData.VT[he]]
			 +			 u * sbtData.VN[sbtData.VT[he + 1]]
			 +			 v * sbtData.VN[sbtData.VT[he + 2]])
		: Ng;

	// ------------------------------------------------------------------
	// face-forward and normalize normals
	// ------------------------------------------------------------------
	const vertex_cu rayDir = (vertex_cu) optixGetWorldRayDirection();

	if((rayDir , Ng) > 0.f) Ng = -Ng;
	Ng /= *Ng;

	if((Ng , Ns) < 0.f)
		Ns -= 2.f * (Ng , Ns) * Ng;
	Ns /= *Ns;

	// ------------------------------------------------------------------
	// compute diffuse material color, including diffuse texture, if
	// available
	// ------------------------------------------------------------------
	vertex_cu diffuseColor(230.0/255, 240.0/255, 250.0/255);

	// ------------------------------------------------------------------
	// compute shadow
	// ------------------------------------------------------------------
	const vertex_cu surfPos = (1.f - u - v) * A + u * B + v * C;
	const vertex_cu lightPos(-907.108f, 2205.875f, -400.0267f);
	const vertex_cu lightDir = lightPos - surfPos;

	// trace shadow ray:
	vertex_cu lightVisibility = 0.f;
	// the values we store the PRD pointer in:
	uint32_t u0, u1;
	packPointer(&lightVisibility, u0, u1);
	optixTrace(params.traversable,
						 surfPos + 1e-3f * Ng,
						 lightDir,
						 1e-3f,			// tmin
						 1.f-1e-3f,	// tmax
						 0.0f,			 // rayTime
						 OptixVisibilityMask(255),
						 // For shadow rays: skip any/closest hit shaders and terminate on first
						 // intersection with anything. The miss shader is used to mark if the
						 // light was visible.
						 OPTIX_RAY_FLAG_DISABLE_ANYHIT
						 | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
						 | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
						 1,						// SBT offset
						 2,							 // SBT stride
						 1,						// missSBTIndex
						 u0, u1);

	// ------------------------------------------------------------------
	// final shading: a bit of ambient, a bit of directional ambient,
	// and directional component based on shadowing
	// ------------------------------------------------------------------
	const float cosDN = 0.1f + .8f * fabsf((rayDir , Ns));

	vertex_cu & prd = *(vertex_cu * ) getPRD<vertex_cu>();
	prd = (.1f + (.2f + .8f * lightVisibility) * cosDN) * diffuseColor;
}

extern "C" __global__ void __anyhit__radiance() {}

extern "C" __global__ void __anyhit__shadow() {}

//------------------------------------------------------------------------------
// miss program that gets called for any ray that did not have a
// valid intersection
//
// as with the anyhit/closest hit programs, in this example we only
// need to have _some_ dummy function to set up a valid SBT
// ------------------------------------------------------------------------------

extern "C" __global__ void __miss__radiance()
{
	vertex_cu &prd = *(vertex_cu *) getPRD<vertex_cu>();
	// set to constant white as background color
	prd = vertex_cu(1.f);
}

extern "C" __global__ void __miss__shadow()
{
	// we didn't hit anything, so the light is visible
	vertex_cu &prd = *(vertex_cu *)getPRD<vertex_cu>();
	prd = vertex_cu(1.f);
}

//------------------------------------------------------------------------------
// ray gen program - the actual rendering happens in here
//------------------------------------------------------------------------------
extern "C" __global__ void __raygen__render_frame()
{
	// compute a test pattern based on pixel ID
	const int ix = optixGetLaunchIndex().x;
	const int iy = optixGetLaunchIndex().y;

	const auto &camera = params.camera;

	// our per-ray data for this example. what we initialize it to
	// won't matter, since this value will be overwritten by either
	// the miss or hit program, anyway
	vertex_cu pixelColorPRD = vertex_cu(0.f);

	// the values we store the PRD pointer in:
	uint32_t u0, u1;
	packPointer(&pixelColorPRD, u0, u1);

	// normalized screen plane position, in [0,1]^2
	const float xscreen = (ix + .5f) / params.frame.width;
	const float yscreen = (iy + .5f) / params.frame.height;

	// generate ray direction
	vertex_cu rayDir = camera.direction + (xscreen - 0.5f) * camera.horizontal + (yscreen - 0.5f) * camera.vertical;
	rayDir /= *rayDir;

	optixTrace(params.traversable,
						 camera.position,
						 rayDir,
						 0.f,		// tmin
						 1e20f,	// tmax
						 0.0f,	 // rayTime
						 OptixVisibilityMask(255),
						 OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
						 0,						// SBT offset
						 2,							 // SBT stride
						 0,						// missSBTIndex
						 u0, u1);

	const int r = int(255.99f*pixelColorPRD.x);
	const int g = int(255.99f*pixelColorPRD.y);
	const int b = int(255.99f*pixelColorPRD.z);

	// convert to 32-bit rgba value (we explicitly set alpha to 0xff
	// to make stb_image_write happy ...
	const uint32_t rgba = 0xff000000
		| (r<<0) | (g<<8) | (b<<16);

	// and write to frame buffer ...
	const uint32_t fbIndex = ix + iy * params.frame.width;
	params.frame.colorBuffer[fbIndex] = rgba;
}


} // namespace gproshan

#endif // GPROSHAN_OPTIX

