#include "include_arma.h"

#include <cassert>

#include <hipsolver.h>
#include <hipsolver.h>


// geometry processing and shape analysis framework
namespace gproshan {


struct cu_spAxb
{
	int * A_col_ptrs, * A_row_indices;
	real_t * A_values, * x, * b;

	cu_spAxb(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb)
	{
		hipMalloc(&A_col_ptrs, (m + 1) * sizeof(int));
		hipMemcpy(A_col_ptrs, hA_col_ptrs, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

		hipMalloc(&A_row_indices, nnz * sizeof(int));
		hipMemcpy(A_row_indices, hA_row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);

		hipMalloc(&A_values, nnz * sizeof(real_t));
		hipMemcpy(A_values, hA_values, nnz * sizeof(real_t), hipMemcpyHostToDevice);

		hipMalloc(&b, nnz * sizeof(real_t));
		hipMemcpy(b, hb, nnz * sizeof(real_t), hipMemcpyHostToDevice);

		hipMalloc(&x, m * sizeof(real_t));
	}

	~cu_spAxb()
	{
		hipFree(A_col_ptrs);
		hipFree(A_row_indices);
		hipFree(A_values);
		hipFree(b);
		hipFree(x);
	}
};

double solve_positive_definite_cusolver(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx, const bool host)
{
	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// solve Ax = b

	int singularity;

	hipsolverSpHandle_t handle_cusolver;
	hipsolverSpCreate(&handle_cusolver);

	hipsparseMatDescr_t descr = 0;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	if(host)
	{
		#ifdef GPROSHAN_FLOAT
			hipsolverSpScsrlsvcholHost(handle_cusolver, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, hb, 0, 0, hx, &singularity);
		#else
			hipsolverSpDcsrlsvcholHost(handle_cusolver, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, hb, 0, 0, hx, &singularity);
		#endif
	}
	else
	{
		// allocate A, x, b into device
		cu_spAxb data(m, nnz, hA_values, hA_col_ptrs, hA_row_indices, hb);

		hipsolverStatus_t status;
		#ifdef GPROSHAN_FLOAT
			status = hipsolverSpScsrlsvchol(handle_cusolver, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, data.b, 0, 0, data.x, &singularity);
		#else
			status = hipsolverSpDcsrlsvchol(handle_cusolver, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, data.b, 0, 0, data.x, &singularity);
		#endif

		if(status == HIPSOLVER_STATUS_SUCCESS)
			hipMemcpy(hx, data.x, m * sizeof(real_t), hipMemcpyDeviceToHost);
		else
			memset(hx, 0, m * sizeof(real_t));
	}

//	printf("%d\n", singularity != -1);

	hipsparseDestroyMatDescr(descr);
	hipsolverSpDestroy(handle_cusolver);

	// end Ax = b

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (double) time / 1000;
}

double solve_positive_definite_cusolver_preview(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx, const bool host)
{
	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// SOLVE Ax = b

	hipsolverSpHandle_t cusolver_handle = nullptr;
	hipsparseHandle_t cusparse_handle = nullptr;
//	hipStream_t stream = nullptr;

	hipsparseMatDescr_t descr = nullptr;

	size_t size_iternal = 0;
	size_t size_chol = 0;

	void * buffer = nullptr;

	int singularity;

	hipsolverSpCreate(&cusolver_handle);
	hipsparseCreate(&cusparse_handle);
/*
	hipStreamCreate(&stream);
	hipsolverSpSetStream(cusolver_handle, stream);
	hipsparseSetStream(cusparse_handle, stream);
*/
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);


	if(host)
	{
		csrcholInfoHost_t info;
		cusolverSpCreateCsrcholInfoHost(&info);

		cusolverSpXcsrcholAnalysisHost(cusolver_handle, m, nnz, descr, hA_col_ptrs, hA_row_indices, info);

		#ifdef GPROSHAN_FLOAT
			cusolverSpScsrcholBufferInfoHost(cusolver_handle, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, info, &size_iternal, &size_chol);
		#else
			cusolverSpDcsrcholBufferInfoHost(cusolver_handle, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, info, &size_iternal, &size_chol);
		#endif

		buffer = new char[size_chol];

		#ifdef GPROSHAN_FLOAT
			cusolverSpScsrcholFactorHost(cusolver_handle, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, info, buffer);
			cusolverSpScsrcholZeroPivotHost(cusolver_handle, info, 0, &singularity);
		#else
			cusolverSpDcsrcholFactorHost(cusolver_handle, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, info, buffer);
			cusolverSpDcsrcholZeroPivotHost(cusolver_handle, info, 0, &singularity);
		#endif
		assert(singularity == -1);

		// SOLVE
		hipEventRecord(start, 0);

		#ifdef GPROSHAN_FLOAT
			cusolverSpScsrcholSolveHost(cusolver_handle, m, hb, hx, info, buffer);
		#else
			cusolverSpDcsrcholSolveHost(cusolver_handle, m, hb, hx, info, buffer);
		#endif

		// END SOLVE
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);

		// FREE
		delete [] (char*) buffer;
		cusolverSpDestroyCsrcholInfoHost(info);
	}
	else
	{
		cu_spAxb data(m, nnz, hA_values, hA_col_ptrs, hA_row_indices, hb);

		csrcholInfo_t info;
		cusolverSpCreateCsrcholInfo(&info);

		cusolverSpXcsrcholAnalysis(cusolver_handle, m, nnz, descr, data.A_col_ptrs, data.A_row_indices, info);

		#ifdef GPROSHAN_FLOAT
			cusolverSpScsrcholBufferInfo(cusolver_handle, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, info, &size_iternal, &size_chol);
		#else
			cusolverSpDcsrcholBufferInfo(cusolver_handle, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, info, &size_iternal, &size_chol);
		#endif

		hipMalloc(&buffer, size_chol);

		#ifdef GPROSHAN_FLOAT
			cusolverSpScsrcholFactor(cusolver_handle, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, info, buffer);
			cusolverSpScsrcholZeroPivot(cusolver_handle, info, 0, &singularity);
		#else
			cusolverSpDcsrcholFactor(cusolver_handle, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, info, buffer);
			cusolverSpDcsrcholZeroPivot(cusolver_handle, info, 0, &singularity);
		#endif

//		assert(singularity == -1);

		// SOLVE
		hipEventRecord(start, 0);

		#ifdef GPROSHAN_FLOAT
			cusolverSpScsrcholSolve(cusolver_handle, m, data.b, data.x, info, buffer);
		#else
			cusolverSpDcsrcholSolve(cusolver_handle, m, data.b, data.x, info, buffer);
		#endif

		// END SOLVE
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);

		hipMemcpy(hx, data.x, m * sizeof(real_t), hipMemcpyDeviceToHost);

		// FREE
		hipFree(buffer);
		cusolverSpDestroyCsrcholInfo(info);
	}

//	hipStreamDestroy(stream);
	hipsparseDestroyMatDescr(descr);
	hipsparseDestroy(cusparse_handle);
	hipsolverSpDestroy(cusolver_handle);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (double) time / 1000;
}


} // namespace gproshan

