#include "hip/hip_runtime.h"
#include "include_arma.h"

#include <cassert>

#include <hipsolver.h>
#include <hipsolver.h>


// geometry processing and shape analysis framework
namespace gproshan {


struct cu_spAxb
{
	int * A_col_ptrs, * A_row_indices;
	real_t * A_values, * x, * b;
	
	cu_spAxb(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx)
	{
		hipMalloc(&A_col_ptrs, (m + 1) * sizeof(int));
		hipMemcpy(A_col_ptrs, hA_col_ptrs, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

		hipMalloc(&A_row_indices, nnz * sizeof(int));
		hipMemcpy(A_row_indices, hA_row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);

		hipMalloc(&A_values, nnz * sizeof(real_t));
		hipMemcpy(A_values, hA_values, nnz * sizeof(real_t), hipMemcpyHostToDevice); 
		
		hipMalloc(&b, nnz * sizeof(real_t));
		hipMemcpy(b, hb, nnz * sizeof(real_t), hipMemcpyHostToDevice);

		hipMalloc(&x, m * sizeof(real_t));
	}

	~cu_spAxb()
	{
		hipFree(A_col_ptrs);
		hipFree(A_row_indices);
		hipFree(A_values);
		hipFree(b);
		hipFree(x);
	}
};

double solve_positive_definite_cusolver(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx, const bool host)
{
	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	// solve Ax = b

	int singularity;
	
	hipsolverSpHandle_t handle_cusolver;
	hipsolverSpCreate(&handle_cusolver);

	hipsparseMatDescr_t descr = 0;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
	
	if(host)
	{
		#ifdef SINGLE_P
			hipsolverSpScsrlsvcholHost(handle_cusolver, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, hb, 0, 0, hx, &singularity);
		#else
			hipsolverSpDcsrlsvcholHost(handle_cusolver, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, hb, 0, 0, hx, &singularity);
		#endif
	}
	else
	{
		// allocate A, x, b into device
		cu_spAxb data(m, nnz, hA_values, hA_col_ptrs, hA_row_indices, hb, hx);

		hipsolverStatus_t status;
		#ifdef SINGLE_P
			status = hipsolverSpScsrlsvchol(handle_cusolver, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, data.b, 0, 0, data.x, &singularity);
		#else
			status = hipsolverSpDcsrlsvchol(handle_cusolver, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, data.b, 0, 0, data.x, &singularity);
		#endif

		if(status == HIPSOLVER_STATUS_SUCCESS)
			hipMemcpy(hx, data.x, m * sizeof(real_t), hipMemcpyDeviceToHost);
		else
			memset(hx, 0, m * sizeof(real_t));
	}

//	printf("%d\n", singularity != -1);

	hipsparseDestroyMatDescr(descr);
	hipsolverSpDestroy(handle_cusolver);

	// end Ax = b

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (double) time / 1000;
}

double solve_positive_definite_cusparse(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx)
{
	hipDeviceReset();
	
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate A, x, b into device
	cu_spAxb data(m, nnz, hA_values, hA_col_ptrs, hA_row_indices, hb, hx);
	
	// aux vector y to device
	real_t * dy;
	hipMalloc(&dy, m * sizeof(real_t));
	
	hipsparseHandle_t handle;
	hipsparseCreate(&handle);

	// SOLVE Ax = b
	
	hipsparseMatDescr_t descr_M = 0;
	hipsparseMatDescr_t descr_L = 0;
	
	csric02Info_t info_M = 0;
	csrsv2Info_t info_L = 0;
	csrsv2Info_t info_Lt = 0;
	
	int buffer_size_M;
	int buffer_size_L;
	int buffer_size_Lt;
	int buffer_size;
	
	void * buffer = 0;

	int structural_zero;
	int numerical_zero;

	const real_t alpha = 1.;
	const hipsparseSolvePolicy_t policy_M = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
	const hipsparseSolvePolicy_t policy_L = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
	const hipsparseSolvePolicy_t policy_Lt = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
	const hipsparseOperation_t trans_L = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	const hipsparseOperation_t trans_Lt = HIPSPARSE_OPERATION_TRANSPOSE;

	hipsparseCreateMatDescr(&descr_M);
	hipsparseSetMatIndexBase(descr_M, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);

	hipsparseCreateMatDescr(&descr_L);
	hipsparseSetMatIndexBase(descr_L, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT);

	hipsparseCreateCsric02Info(&info_M);
	hipsparseCreateCsrsv2Info(&info_L);
	hipsparseCreateCsrsv2Info(&info_Lt);

	#ifdef SINGLE_P
		hipsparseScsric02_bufferSize(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, &buffer_size_M);
		hipsparseScsrsv2_bufferSize(handle, trans_L, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, &buffer_size_L);
		hipsparseScsrsv2_bufferSize(handle, trans_Lt, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, &buffer_size_Lt);
	#else
		hipsparseDcsric02_bufferSize(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, &buffer_size_M);
		hipsparseDcsrsv2_bufferSize(handle, trans_L, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, &buffer_size_L);
		hipsparseDcsrsv2_bufferSize(handle, trans_Lt, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, &buffer_size_Lt);
	#endif

	buffer_size = max(buffer_size_M, max(buffer_size_L, buffer_size_Lt));
	hipMalloc(&buffer, buffer_size);

	#ifdef SINGLE_P
		hipsparseScsric02_analysis(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, policy_M, buffer);
	#else
		hipsparseDcsric02_analysis(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, policy_M, buffer);
	#endif
	if(HIPSPARSE_STATUS_ZERO_PIVOT == hipsparseXcsric02_zeroPivot(handle, info_M, &structural_zero))
		printf("A(%d,%d) is missing\n", structural_zero, structural_zero);

	#ifdef SINGLE_P
		hipsparseScsrsv2_analysis(handle, trans_L, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, policy_L, buffer);
		hipsparseScsrsv2_analysis(handle, trans_Lt, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, policy_Lt, buffer);

		hipsparseScsric02(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, policy_M, buffer);
	#else
		hipsparseDcsrsv2_analysis(handle, trans_L, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, policy_L, buffer);
		hipsparseDcsrsv2_analysis(handle, trans_Lt, m, nnz, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, policy_Lt, buffer);

		hipsparseDcsric02(handle, m, nnz, descr_M, data.A_values, data.A_col_ptrs, data.A_row_indices, info_M, policy_M, buffer);
	#endif
	if(HIPSPARSE_STATUS_ZERO_PIVOT == hipsparseXcsric02_zeroPivot(handle, info_M, &numerical_zero))
		printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);


	// SOLVE
	hipEventRecord(start, 0);
	
	#ifdef SINGLE_P
		hipsparseScsrsv2_solve(handle, trans_L, m, nnz, &alpha, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, data.b, dy, policy_L, buffer);
		hipsparseScsrsv2_solve(handle, trans_Lt, m, nnz, &alpha, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, dy, data.x, policy_Lt, buffer);
	#else
		hipsparseDcsrsv2_solve(handle, trans_L, m, nnz, &alpha, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_L, data.b, dy, policy_L, buffer);
		hipsparseDcsrsv2_solve(handle, trans_Lt, m, nnz, &alpha, descr_L, data.A_values, data.A_col_ptrs, data.A_row_indices, info_Lt, dy, data.x, policy_Lt, buffer);
	#endif
	
	// copy sol x to host
	hipMemcpy(hx, data.x, m * sizeof(real_t), hipMemcpyDeviceToHost);
	
	// END SOLVE
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	
	// FREE
	hipFree(buffer);
	hipsparseDestroyMatDescr(descr_M);
	hipsparseDestroyMatDescr(descr_L);
	hipsparseDestroyCsric02Info(info_M);
	hipsparseDestroyCsrsv2Info(info_L);
	hipsparseDestroyCsrsv2Info(info_Lt);
	hipsparseDestroy(handle);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (double) time / 1000;
}

double solve_positive_definite_cusolver_preview(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx, const bool host)
{
	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// SOLVE Ax = b

	hipsolverSpHandle_t cusolver_handle = nullptr;
	hipsparseHandle_t cusparse_handle = nullptr;
//	hipStream_t stream = nullptr;

	hipsparseMatDescr_t descr = nullptr;

	size_t size_iternal = 0;
	size_t size_chol = 0;

	void * buffer = nullptr;

	int singularity;

	hipsolverSpCreate(&cusolver_handle);
	hipsparseCreate(&cusparse_handle);
/*
	hipStreamCreate(&stream);
	hipsolverSpSetStream(cusolver_handle, stream);
	hipsparseSetStream(cusparse_handle, stream);
*/	
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	
	if(host)
	{
		csrcholInfoHost_t info;
		cusolverSpCreateCsrcholInfoHost(&info);

		cusolverSpXcsrcholAnalysisHost(cusolver_handle, m, nnz, descr, hA_col_ptrs, hA_row_indices, info);

		#ifdef SINGLE_P
			cusolverSpScsrcholBufferInfoHost(cusolver_handle, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, info, &size_iternal, &size_chol);
		#else
			cusolverSpDcsrcholBufferInfoHost(cusolver_handle, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, info, &size_iternal, &size_chol);
		#endif
	
		buffer = new char[size_chol];
		
		#ifdef SINGLE_P
			cusolverSpScsrcholFactorHost(cusolver_handle, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, info, buffer);
			cusolverSpScsrcholZeroPivotHost(cusolver_handle, info, 0, &singularity);
		#else
			cusolverSpDcsrcholFactorHost(cusolver_handle, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, info, buffer);
			cusolverSpDcsrcholZeroPivotHost(cusolver_handle, info, 0, &singularity);
		#endif
		assert(singularity == -1);

		// SOLVE
		hipEventRecord(start, 0);
		
		#ifdef SINGLE_P
			cusolverSpScsrcholSolveHost(cusolver_handle, m, hb, hx, info, buffer);
		#else
			cusolverSpDcsrcholSolveHost(cusolver_handle, m, hb, hx, info, buffer);
		#endif
		
		// END SOLVE
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);

		// FREE
		delete [] (char*) buffer;
		cusolverSpDestroyCsrcholInfoHost(info);
	}
	else
	{
		cu_spAxb data(m, nnz, hA_values, hA_col_ptrs, hA_row_indices, hb, hx);

		csrcholInfo_t info;
		cusolverSpCreateCsrcholInfo(&info);

		cusolverSpXcsrcholAnalysis(cusolver_handle, m, nnz, descr, data.A_col_ptrs, data.A_row_indices, info);

		#ifdef SINGLE_P
			cusolverSpScsrcholBufferInfo(cusolver_handle, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, info, &size_iternal, &size_chol);
		#else
			cusolverSpDcsrcholBufferInfo(cusolver_handle, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, info, &size_iternal, &size_chol);
		#endif
	
		hipMalloc(&buffer, size_chol);

		#ifdef SINGLE_P
			cusolverSpScsrcholFactor(cusolver_handle, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, info, buffer);
			cusolverSpScsrcholZeroPivot(cusolver_handle, info, 0, &singularity);
		#else
			cusolverSpDcsrcholFactor(cusolver_handle, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, info, buffer);
			cusolverSpDcsrcholZeroPivot(cusolver_handle, info, 0, &singularity);
		#endif

//		assert(singularity == -1);

		// SOLVE
		hipEventRecord(start, 0);
		
		#ifdef SINGLE_P
			cusolverSpScsrcholSolve(cusolver_handle, m, data.b, data.x, info, buffer);
		#else
			cusolverSpDcsrcholSolve(cusolver_handle, m, data.b, data.x, info, buffer);
		#endif
		
		// END SOLVE
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);

		hipMemcpy(hx, data.x, m * sizeof(real_t), hipMemcpyDeviceToHost);

		// FREE
		hipFree(buffer);
		cusolverSpDestroyCsrcholInfo(info);
	}

//	hipStreamDestroy(stream);
	hipsparseDestroyMatDescr(descr);
	hipsparseDestroy(cusparse_handle);
	hipsolverSpDestroy(cusolver_handle);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	return (double) time / 1000;
}


} // namespace gproshan

