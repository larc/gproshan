#include "geodesics/che.cuh"


// geometry processing and shape analysis framework
namespace gproshan {


__host__ __device__
index_t cu_trig(index_t he)
{
	if(he == NIL) return NIL;
	return he / che::P;
}

__host__ __device__
index_t cu_next(index_t he)
{
	if(he == NIL) return NIL;
	return che::P * cu_trig(he) + (he + 1) % che::P;
}

__host__ __device__
index_t cu_prev(index_t he)
{
	if(he == NIL) return NIL;
	return che::P * cu_trig(he) + (he + che::P - 1) % che::P;
}

void cuda_create_CHE(CHE * h_che, CHE *& dd_che, CHE *& d_che)
{
	dd_che = (CHE *) malloc(sizeof(CHE));
	memcpy(dd_che, h_che, sizeof(CHE));

	hipMalloc(&dd_che->GT, sizeof(vertex_cu) * h_che->n_vertices);
	hipMemcpy(dd_che->GT, h_che->GT, sizeof(vertex_cu) * h_che->n_vertices, hipMemcpyHostToDevice);

	hipMalloc(&dd_che->VT, sizeof(index_t) * h_che->n_half_edges);
	hipMemcpy(dd_che->VT, h_che->VT, sizeof(index_t) * h_che->n_half_edges, hipMemcpyHostToDevice);

	hipMalloc(&dd_che->OT, sizeof(index_t) * h_che->n_half_edges);
	hipMemcpy(dd_che->OT, h_che->OT, sizeof(index_t) * h_che->n_half_edges, hipMemcpyHostToDevice);

	hipMalloc(&dd_che->EVT, sizeof(index_t) * h_che->n_vertices);
	hipMemcpy(dd_che->EVT, h_che->EVT, sizeof(index_t) * h_che->n_vertices, hipMemcpyHostToDevice);

	hipMalloc(&d_che, sizeof(CHE));
	hipMemcpy(d_che, dd_che, sizeof(CHE), hipMemcpyHostToDevice);
}

void cuda_free_CHE(CHE *& dd_che, CHE *& d_che)
{
	if(dd_che->GT) hipFree(dd_che->GT);
	if(dd_che->VT) hipFree(dd_che->VT);
	if(dd_che->OT) hipFree(dd_che->OT);
	if(dd_che->EVT) hipFree(dd_che->EVT);

	free(dd_che);
	hipFree(d_che);
}


} // namespace gproshan

