#include "mesh/che.cuh"


// geometry processing and shape analysis framework
namespace gproshan {


__host__ __device__
index_t cu_trig(index_t he)
{
	if(he == NIL) return NIL;
	return he / che::mtrig;
}

__host__ __device__
index_t cu_next(index_t he)
{
	if(he == NIL) return NIL;
	return che::mtrig * cu_trig(he) + (he + 1) % che::mtrig;
}

__host__ __device__
index_t cu_prev(index_t he)
{
	if(he == NIL) return NIL;
	return che::mtrig * cu_trig(he) + (he + che::mtrig - 1) % che::mtrig;
}

void cuda_create_CHE(CHE * h_che, CHE *& dd_che, CHE *& d_che, const bool & normal)
{
	dd_che = (CHE *) malloc(sizeof(CHE));
	memcpy(dd_che, h_che, sizeof(CHE));

	hipMalloc(&dd_che->GT, sizeof(vertex_cu) * h_che->n_vertices);
	hipMemcpy(dd_che->GT, h_che->GT, sizeof(vertex_cu) * h_che->n_vertices, hipMemcpyHostToDevice);
	
	if(normal)
	{
		hipMalloc(&dd_che->VN, sizeof(vertex_cu) * h_che->n_vertices);
		hipMemcpy(dd_che->VN, h_che->VN, sizeof(vertex_cu) * h_che->n_vertices, hipMemcpyHostToDevice);
	}

	hipMalloc(&dd_che->VT, sizeof(index_t) * h_che->n_half_edges);
	hipMemcpy(dd_che->VT, h_che->VT, sizeof(index_t) * h_che->n_half_edges, hipMemcpyHostToDevice);

	hipMalloc(&dd_che->OT, sizeof(index_t) * h_che->n_half_edges);
	hipMemcpy(dd_che->OT, h_che->OT, sizeof(index_t) * h_che->n_half_edges, hipMemcpyHostToDevice);

	hipMalloc(&dd_che->EVT, sizeof(index_t) * h_che->n_vertices);
	hipMemcpy(dd_che->EVT, h_che->EVT, sizeof(index_t) * h_che->n_vertices, hipMemcpyHostToDevice);

	hipMalloc(&d_che, sizeof(CHE));
	hipMemcpy(d_che, dd_che, sizeof(CHE), hipMemcpyHostToDevice);
}

void cuda_free_CHE(CHE *& dd_che, CHE *& d_che)
{
	if(dd_che->GT) hipFree(dd_che->GT);
	if(dd_che->VN) hipFree(dd_che->VN);
	if(dd_che->VT) hipFree(dd_che->VT);
	if(dd_che->OT) hipFree(dd_che->OT);
	if(dd_che->EVT) hipFree(dd_che->EVT);

	free(dd_che);
	hipFree(d_che);
}


} // namespace gproshan

