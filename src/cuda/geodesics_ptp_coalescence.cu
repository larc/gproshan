#include "hip/hip_runtime.h"
#include "geodesics_ptp_coalescence.cuh"

#include "che_off.h"

#include <cstdio>
#include <fstream>
#include <cassert>
#include <hipblas.h>

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

using namespace std;


// geometry processing and shape analysis framework
namespace gproshan {


double parallel_toplesets_propagation_coalescence_gpu(const ptp_out_t & ptp_out, const che * mesh, const vector<index_t> & sources, const toplesets_t & toplesets, const bool & set_inf)
{
	index_t * inv = nullptr;
	che * coalescence_mesh = ptp_coalescence(inv, mesh, toplesets);

	// ------------------------------------------------------
	
	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// BEGIN PTP

	CHE * h_mesh = new CHE(coalescence_mesh);
	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	distance_t * h_dist = new distance_t[h_mesh->n_vertices];

	distance_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(distance_t) * h_mesh->n_vertices);
	hipMalloc(&d_dist[1], sizeof(distance_t) * h_mesh->n_vertices);

	distance_t * d_error;
	hipMalloc(&d_error, sizeof(distance_t) * h_mesh->n_vertices);

	index_t d;
	if(ptp_out.clusters)
	{
		index_t * h_clusters = new index_t[h_mesh->n_vertices];
		index_t * d_clusters[2] = {nullptr, nullptr};
		
		hipMalloc(&d_clusters[0], sizeof(index_t) * h_mesh->n_vertices);
		hipMalloc(&d_clusters[1], sizeof(index_t) * h_mesh->n_vertices);

		d = run_ptp_coalescence_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, sources, {toplesets.limits, inv}, d_error, h_clusters, d_clusters);
		hipMemcpy(h_clusters, d_clusters[d], sizeof(index_t) * h_mesh->n_vertices, hipMemcpyDeviceToHost);

		#pragma omp parallel for
		for(index_t i = 0; i < h_mesh->n_vertices; i++)
			ptp_out.clusters[toplesets.index[i]] = h_clusters[i];

		hipFree(d_clusters[0]);
		hipFree(d_clusters[1]);

		delete [] h_clusters;
	}
	else d = run_ptp_coalescence_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, sources, {toplesets.limits, inv}, d_error);

	hipMemcpy(h_dist, d_dist[d], sizeof(distance_t) * h_mesh->n_vertices, hipMemcpyDeviceToHost);

	hipFree(d_error);
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	cuda_free_CHE(dd_mesh, d_mesh);

	delete coalescence_mesh;
	delete [] inv;

	#pragma omp parallel for
	for(index_t i = 0; i < toplesets.limits.back(); i++)
		ptp_out.dist[toplesets.index[i]] = h_dist[i];
	
	delete [] h_dist;

	// END PTP

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return time / 1000;
}

index_t run_ptp_coalescence_gpu(CHE * d_mesh, const index_t & n_vertices, distance_t * h_dist, distance_t ** d_dist, const vector<index_t> & sources, const toplesets_t & inv, distance_t * d_error, index_t * h_clusters, index_t ** d_clusters)
{
	#pragma omp parallel for
	for(index_t v = 0; v < n_vertices; v++)
		h_dist[v] = INFINITY;

	for(index_t i = 0; i < sources.size(); i++)
		h_dist[inv.index[sources[i]]] = 0;

	hipMemcpy(d_dist[0], h_dist, sizeof(distance_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_dist[1], h_dist, sizeof(distance_t) * n_vertices, hipMemcpyHostToDevice);

	if(h_clusters)
	{
		assert(d_clusters[0]);

		for(index_t i = 0; i < sources.size(); i++)
			h_clusters[inv.index[sources[i]]] = i + 1;

		hipMemcpy(d_clusters[0], h_clusters, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);
		hipMemcpy(d_clusters[1], h_clusters, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);
	}

	index_t d = 0;
	index_t start, end, n_cond;
	index_t i = 1, j = 2;

	// maximum number of iterations
	index_t iter = 0;
	index_t max_iter = inv.limits.size() << 1;

	while(i < j && iter++ < max_iter)
	{
		if(i < (j >> 1)) i = (j >> 1); // K/2 limit band size
		
		start = inv.limits[i];
		end = inv.limits[j];
		n_cond = inv.limits[i + 1] - start;

		if(h_clusters)
			relax_ptp_coalescence <<< NB(end - start), NT >>> (d_mesh, d_dist[!d], d_dist[d], d_clusters[!d], d_clusters[d], end, start);
		else
			relax_ptp_coalescence <<< NB(end - start), NT >>> (d_mesh, d_dist[!d], d_dist[d], end, start);
		
		hipDeviceSynchronize();
		
		relative_error <<< NB(n_cond), NT >>>(d_error, d_dist[!d], d_dist[d], start, start + n_cond);
		hipDeviceSynchronize();
		
		if(n_cond == thrust::count_if(thrust::device, d_error + start, d_error + start + n_cond, is_ok()))
			i++;
		
		if(j < inv.limits.size() - 1) j++;
		
		d = !d;
	}

	return d;
}

__global__
void relax_ptp_coalescence(CHE * mesh, distance_t * new_dist, distance_t * old_dist, index_t end, index_t start)
{
	index_t v = blockDim.x * blockIdx.x + threadIdx.x + start;

	if(v < end)
	{
		if(v < mesh->n_vertices)
		{
			new_dist[v] = old_dist[v];

			distance_t d;
			cu_for_star(he, mesh, v)
			{
				d = cu_update_step(mesh, old_dist, he);
				if(d < new_dist[v]) new_dist[v] = d;
			}
		}
	}
}


__global__
void relax_ptp_coalescence(CHE * mesh, distance_t * new_dist, distance_t * old_dist, index_t * new_clusters, index_t * old_clusters, index_t end, index_t start)
{
	index_t v = blockDim.x * blockIdx.x + threadIdx.x + start;

	if(v < end)
	{
		if(v < mesh->n_vertices)
		{
			new_dist[v] = old_dist[v];
			new_clusters[v] = old_clusters[v];

			distance_t d;
			cu_for_star(he, mesh, v)
			{
				d = cu_update_step(mesh, old_dist, he);
				if(d < new_dist[v])
				{
					new_dist[v] = d;
					new_clusters[v] = old_dist[mesh->VT[cu_prev(he)]] < old_dist[mesh->VT[cu_next(he)]] ? old_clusters[mesh->VT[cu_prev(he)]] : old_clusters[mesh->VT[cu_next(he)]];
				}
			}
		}
	}
}

__forceinline__ __device__
distance_t cu_update_step(CHE * mesh, const distance_t * dist, const index_t & he)
{
	index_t x[3];
	x[0] = mesh->VT[cu_next(he)];
	x[1] = mesh->VT[cu_prev(he)];
	x[2] = mesh->VT[he];

	vertex_cu X[2];
	X[0] = mesh->GT[x[0]] - mesh->GT[x[2]];
	X[1] = mesh->GT[x[1]] - mesh->GT[x[2]];

	distance_t t[2];
	t[0] = dist[x[0]];
	t[1] = dist[x[1]];

	distance_t q[2][2];
	q[0][0] = (X[0], X[0]);
	q[0][1] = (X[0], X[1]);
	q[1][0] = (X[1], X[0]);
	q[1][1] = (X[1], X[1]);

	distance_t det = q[0][0] * q[1][1] - q[0][1] * q[1][0];
	distance_t Q[2][2];
	Q[0][0] = q[1][1] / det;
	Q[0][1] = -q[0][1] / det;
	Q[1][0] = -q[1][0] / det;
	Q[1][1] = q[0][0] / det;

	distance_t delta = t[0] * (Q[0][0] + Q[1][0]) + t[1] * (Q[0][1] + Q[1][1]);
	distance_t dis = delta * delta - (Q[0][0] + Q[0][1] + Q[1][0] + Q[1][1]) * (t[0]*t[0]*Q[0][0] + t[0]*t[1]*(Q[1][0] + Q[0][1]) + t[1]*t[1]*Q[1][1] - 1);

	distance_t p;

	if(dis >= 0)
	{
		#ifdef SINGLE_P
			p = delta + sqrtf(dis);
		#else
			p = delta + sqrt(dis);
		#endif
		p /= Q[0][0] + Q[0][1] + Q[1][0] + Q[1][1];
	}

	distance_t tp[2];
	tp[0] = t[0] - p;
	tp[1] = t[1] - p;

	vertex_cu n(tp[0] * (X[0][0]*Q[0][0] + X[1][0]*Q[1][0]) + tp[1] * (X[0][0]*Q[0][1] + X[1][0]*Q[1][1]),
			 tp[0] * (X[0][1]*Q[0][0] + X[1][1]*Q[1][0]) + tp[1] * (X[0][1]*Q[0][1] + X[1][1]*Q[1][1]),
			 tp[0] * (X[0][2]*Q[0][0] + X[1][2]*Q[1][0]) + tp[1] * (X[0][2]*Q[0][1] + X[1][2]*Q[1][1]) );

	distance_t cond[2];
	cond[0] = (X[0] , n);
	cond[1] = (X[1] , n);

	distance_t c[2];
	c[0] = cond[0] * Q[0][0] + cond[1] * Q[0][1];
	c[1] = cond[0] * Q[1][0] + cond[1] * Q[1][1];

	if(t[0] == INFINITY || t[1] == INFINITY || dis < 0 || c[0] >= 0 || c[1] >= 0)
	{
		distance_t dp[2];
		dp[0] = dist[x[0]] + *X[0];
		dp[1] = dist[x[1]] + *X[1];

		p = dp[dp[1] < dp[0]];
	}

	return p;
}


} // namespace gproshan

