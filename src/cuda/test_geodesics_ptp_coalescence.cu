#include "test_geodesics_ptp_coalescence.cuh"

#include "geodesics_ptp_coalescence.cuh"
#include "geodesics_ptp.h"
#include "test_geodesics_ptp.h"

#include "che_off.h"

#include <fstream>
#include <hipblas.h>

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>

using namespace std;


// geometry processing and shape analysis framework
namespace gproshan {


vector<pair<index_t, real_t> > iter_error_parallel_toplesets_propagation_coalescence_gpu(che * mesh, const vector<index_t> & sources, const vector<index_t> & limits, const index_t * sorted_index, const real_t * exact_dist, double & time_ptp)
{
	// sort data by levels, must be improve the coalescence

	vertex * V = new vertex[mesh->n_vertices()];
	index_t * F = new index_t[mesh->n_faces() * che::P];
	index_t * inv = new index_t[mesh->n_vertices()];
	real_t * exact_dist_sorted = new real_t[mesh->n_vertices()];
	
	#pragma omp parallel for
	for(index_t i = 0; i < mesh->n_vertices(); i++)
	{
		V[i] = mesh->gt(sorted_index[i]);
		inv[sorted_index[i]] = i;
		exact_dist_sorted[i] = exact_dist[sorted_index[i]];
	}

	#pragma omp parallel for
	for(index_t he = 0; he < mesh->n_half_edges(); he++)
		F[he] = inv[mesh->vt(he)];

	mesh = new che(V, mesh->n_vertices(), F, mesh->n_faces());

	delete [] V;
	delete [] F;

	// ------------------------------------------------------

	hipDeviceReset();
	
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	// BEGIN PTP

	CHE * h_mesh = new CHE(mesh);
	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	real_t * h_dist = new real_t[h_mesh->n_vertices];

	real_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(real_t) * h_mesh->n_vertices);
	hipMalloc(&d_dist[1], sizeof(real_t) * h_mesh->n_vertices);

	real_t * d_error;
	hipMalloc(&d_error, sizeof(real_t) * h_mesh->n_vertices);

	vector<pair<index_t, real_t> > iter_error = iter_error_run_ptp_coalescence_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, sources, limits, inv, exact_dist_sorted, d_error);
	
	delete [] h_dist;
	hipFree(d_error);
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	cuda_free_CHE(dd_mesh, d_mesh);

	// END PTP
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	time_ptp = time / 1000;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	delete mesh;
	delete [] inv;

	return iter_error;
}

/// Return an array of time in seconds.
double * times_farthest_point_sampling_ptp_coalescence_gpu(che * mesh, vector<index_t> & samples, size_t n, real_t radio)
{
	hipDeviceReset();

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// BEGIN FPS PTP
	
	vertex * V = new vertex[mesh->n_vertices()];
	index_t * F = new index_t[mesh->n_faces() * che::P];
	index_t * inv = new index_t[mesh->n_vertices()];


	real_t * h_dist = new real_t[mesh->n_vertices()];

	real_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(real_t) * mesh->n_vertices());
	hipMalloc(&d_dist[1], sizeof(real_t) * mesh->n_vertices());

	real_t * d_error;
	hipMalloc(&d_error, sizeof(real_t) * mesh->n_vertices());

	vector<index_t> limits;
	index_t * toplesets = new index_t[mesh->n_vertices()];
	index_t * sorted_index = new index_t[mesh->n_vertices()];

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	if(n >= mesh->n_vertices()) n = mesh->n_vertices() >> 1;

	double * times = new double[n + 1];

	n -= samples.size();
	samples.reserve(n);

	float time_fps;
	index_t d;
	int f;
	real_t max_dist = INFINITY;
	while(n-- && max_dist > radio)
	{
		hipEventRecord(start, 0);
		
		limits.clear();
		mesh->compute_toplesets(toplesets, sorted_index, limits, samples);
		
		// sort data by levels, must be improve the coalescence
	
		#pragma omp parallel for
		for(index_t i = 0; i < mesh->n_vertices(); i++)
		{
			V[i] = mesh->gt(sorted_index[i]);
			inv[sorted_index[i]] = i;
		}

		#pragma omp parallel for
		for(index_t he = 0; he < mesh->n_half_edges(); he++)
			F[he] = inv[mesh->vt(he)];

		che * tmp_mesh = new che(V, mesh->n_vertices(), F, mesh->n_faces());

		CHE * h_mesh = new CHE(tmp_mesh);
		CHE * dd_mesh, * d_mesh;
		cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

		// exec algorithm
		d = run_ptp_coalescence_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, samples, {limits, inv}, d_error);

		// free memory
		cuda_free_CHE(dd_mesh, d_mesh);
		delete tmp_mesh;

		// 1 indexing
		#ifdef SINGLE_P
			hipblasIsamax(handle, mesh->n_vertices(), d_dist[d], 1, &f);
		#else
			hipblasIdamax(handle, mesh->n_vertices(), d_dist[d], 1, &f);
		#endif
		
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_fps, start, stop);

		times[samples.size()] = time_fps / 1000;

		if(radio > 0 || !n)
			hipMemcpy(&max_dist, d_dist[d] + f - 1, sizeof(real_t), hipMemcpyDeviceToHost);
		
		samples.push_back(sorted_index[f - 1]);
	}

	hipblasDestroy(handle);
	
	delete [] V;
	delete [] F;
	delete [] inv;
	delete [] h_dist;
	delete [] toplesets;
	delete [] sorted_index;

	hipFree(d_error);
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);

	// END FPS PTP

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return times;
}

vector<pair<index_t, real_t> > iter_error_run_ptp_coalescence_gpu(CHE * d_mesh, const index_t & n_vertices, real_t * h_dist, real_t ** d_dist, const vector<index_t> & sources, const vector<index_t> & limits, const index_t * inv, const real_t * exact_dist, real_t * d_error)
{
	#pragma omp parallel for
	for(index_t v = 0; v < n_vertices; v++)
		h_dist[v] = INFINITY;

	for(index_t i = 0; i < sources.size(); i++)
		h_dist[inv[sources[i]]] = 0;

	hipMemcpy(d_dist[0], h_dist, sizeof(real_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_dist[1], h_dist, sizeof(real_t) * n_vertices, hipMemcpyHostToDevice);

	vector<pair<index_t, real_t> > iter_error;
	iter_error.reserve(limits.size());

	ofstream os("band");
	
	index_t d = 0;
	index_t start, end, n_cond;
	index_t i = 1, j = 2;
	index_t n_iter = 0;

	while(i < j)
	{
		n_iter++;
		start = limits[i];
		end = limits[j];
		n_cond = limits[i + 1] - start;

		relax_ptp_coalescence <<< NB(end - start), NT >>> (d_mesh, d_dist[!d], d_dist[d], end, start);
		// print band info
		os << n_iter << " " << i << " " << j << " " << end - start << endl;

		// begin calculating iteration error
		hipMemcpy(h_dist, d_dist[!d], sizeof(real_t) * n_vertices, hipMemcpyDeviceToHost);
		if(j == limits.size() - 1)
			iter_error.push_back(make_pair(n_iter, compute_error(h_dist, exact_dist, n_vertices, sources.size())));
		// end

		relative_error <<< NB(n_cond), NT >>> (d_error, d_dist[!d], d_dist[d], start, start + n_cond);
		hipDeviceSynchronize();
		
		if(n_cond == thrust::count_if(thrust::device, d_error + start, d_error + start + n_cond, is_ok()))
			i++;
		if(j < limits.size() - 1) j++;	
		
		d = !d;
	}

	os.close();

	return iter_error;
}


} // namespace gproshan

