
#include <hip/hip_runtime.h>
#include <cassert>

#include <hipsolver.h>


// geometry processing and shape analysis framework
namespace gproshan {


struct cu_spAxb
{
	int * A_col_ptrs, * A_row_indices;
	double * A_values, * x, * b;

	cu_spAxb(const int m, const int nnz, const double * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const double * hb)
	{
		hipMalloc(&A_col_ptrs, (m + 1) * sizeof(int));
		hipMemcpy(A_col_ptrs, hA_col_ptrs, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

		hipMalloc(&A_row_indices, nnz * sizeof(int));
		hipMemcpy(A_row_indices, hA_row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);

		hipMalloc(&A_values, nnz * sizeof(double));
		hipMemcpy(A_values, hA_values, nnz * sizeof(double), hipMemcpyHostToDevice);

		hipMalloc(&b, nnz * sizeof(double));
		hipMemcpy(b, hb, nnz * sizeof(double), hipMemcpyHostToDevice);

		hipMalloc(&x, m * sizeof(double));
	}

	~cu_spAxb()
	{
		hipFree(A_col_ptrs);
		hipFree(A_row_indices);
		hipFree(A_values);
		hipFree(b);
		hipFree(x);
	}
};

double solve_positive_definite_cusolver(const int m, const int nnz, const double * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const double * hb, double * hx, const bool host)
{
	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// solve Ax = b

	int singularity;

	hipsolverSpHandle_t handle_cusolver;
	hipsolverSpCreate(&handle_cusolver);

	hipsparseMatDescr_t descr = 0;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	if(host)
	{
		hipsolverSpDcsrlsvcholHost(handle_cusolver, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, hb, 0, 0, hx, &singularity);
	}
	else
	{
		// allocate A, x, b into device
		cu_spAxb data(m, nnz, hA_values, hA_col_ptrs, hA_row_indices, hb);

		hipsolverStatus_t status = hipsolverSpDcsrlsvchol(handle_cusolver, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, data.b, 0, 0, data.x, &singularity);

		if(status == HIPSOLVER_STATUS_SUCCESS)
			hipMemcpy(hx, data.x, m * sizeof(double), hipMemcpyDeviceToHost);
		else
			memset(hx, 0, m * sizeof(double));
	}

//	printf("%d\n", singularity != -1);

	hipsparseDestroyMatDescr(descr);
	hipsolverSpDestroy(handle_cusolver);

	// end Ax = b

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (double) time / 1000;
}


} // namespace gproshan

