#include <gproshan/include_arma.h>

#include <cassert>

#include <hipsolver.h>


// geometry processing and shape analysis framework
namespace gproshan {


struct cu_spAxb
{
	int * A_col_ptrs, * A_row_indices;
	real_t * A_values, * x, * b;

	cu_spAxb(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb)
	{
		hipMalloc(&A_col_ptrs, (m + 1) * sizeof(int));
		hipMemcpy(A_col_ptrs, hA_col_ptrs, (m + 1) * sizeof(int), hipMemcpyHostToDevice);

		hipMalloc(&A_row_indices, nnz * sizeof(int));
		hipMemcpy(A_row_indices, hA_row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);

		hipMalloc(&A_values, nnz * sizeof(real_t));
		hipMemcpy(A_values, hA_values, nnz * sizeof(real_t), hipMemcpyHostToDevice);

		hipMalloc(&b, nnz * sizeof(real_t));
		hipMemcpy(b, hb, nnz * sizeof(real_t), hipMemcpyHostToDevice);

		hipMalloc(&x, m * sizeof(real_t));
	}

	~cu_spAxb()
	{
		hipFree(A_col_ptrs);
		hipFree(A_row_indices);
		hipFree(A_values);
		hipFree(b);
		hipFree(x);
	}
};

double solve_positive_definite_cusolver(const int m, const int nnz, const real_t * hA_values, const int * hA_col_ptrs, const int * hA_row_indices, const real_t * hb, real_t * hx, const bool host)
{
	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// solve Ax = b

	int singularity;

	hipsolverSpHandle_t handle_cusolver;
	hipsolverSpCreate(&handle_cusolver);

	hipsparseMatDescr_t descr = 0;
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	if(host)
	{
		#ifdef GPROSHAN_FLOAT
			hipsolverSpScsrlsvcholHost(handle_cusolver, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, hb, 0, 0, hx, &singularity);
		#else
			hipsolverSpDcsrlsvcholHost(handle_cusolver, m, nnz, descr, hA_values, hA_col_ptrs, hA_row_indices, hb, 0, 0, hx, &singularity);
		#endif
	}
	else
	{
		// allocate A, x, b into device
		cu_spAxb data(m, nnz, hA_values, hA_col_ptrs, hA_row_indices, hb);

		hipsolverStatus_t status;
		#ifdef GPROSHAN_FLOAT
			status = hipsolverSpScsrlsvchol(handle_cusolver, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, data.b, 0, 0, data.x, &singularity);
		#else
			status = hipsolverSpDcsrlsvchol(handle_cusolver, m, nnz, descr, data.A_values, data.A_col_ptrs, data.A_row_indices, data.b, 0, 0, data.x, &singularity);
		#endif

		if(status == HIPSOLVER_STATUS_SUCCESS)
			hipMemcpy(hx, data.x, m * sizeof(real_t), hipMemcpyDeviceToHost);
		else
			memset(hx, 0, m * sizeof(real_t));
	}

//	printf("%d\n", singularity != -1);

	hipsparseDestroyMatDescr(descr);
	hipsolverSpDestroy(handle_cusolver);

	// end Ax = b

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (double) time / 1000;
}


} // namespace gproshan

