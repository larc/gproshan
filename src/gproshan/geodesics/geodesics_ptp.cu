#include "hip/hip_runtime.h"
#include <gproshan/geodesics/geodesics_ptp.cuh>
#include <gproshan/geodesics/geodesics_ptp.h>

#include <cstdio>
#include <fstream>
#include <cassert>
#include <hipblas.h>

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>


// geometry processing and shape analysis framework
namespace gproshan {


double parallel_toplesets_propagation_gpu(const ptp_out_t & ptp_out, const che * mesh, const std::vector<index_t> & sources, const toplesets_t & toplesets, const bool & coalescence, const bool & set_inf)
{
	CHE h_mesh(mesh);
	const size_t & n_vertices = h_mesh.n_vertices;

	index_t * inv = nullptr;
	if(coalescence)
	{
		inv = new index_t[n_vertices];
		h_mesh.GT = new vertex[n_vertices];
		h_mesh.EVT = new index_t[n_vertices];
		h_mesh.VT = new index_t[h_mesh.n_half_edges];

		#pragma omp parallel for
		for(index_t i = 0; i < toplesets.limits.back(); ++i)
		{
			h_mesh.GT[i] = mesh->point(toplesets.index[i]);
			inv[toplesets.index[i]] = i;
		}

		#pragma omp parallel for
		for(index_t he = 0; he < mesh->n_half_edges; ++he)
		{
			const index_t & v = mesh->halfedge(he);
			if(v != NIL)
			{
				h_mesh.VT[he] = inv[v];
				if(mesh->evt(v) == he)
					h_mesh.EVT[inv[v]] = he;
			}
		}
	}

	hipDeviceReset();

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(&h_mesh, dd_mesh, d_mesh);

	real_t * h_dist = coalescence ? new real_t[h_mesh.n_vertices] : ptp_out.dist;

	if(set_inf)
	{
		#pragma omp parallel for
		for(index_t v = 0; v < h_mesh.n_vertices; ++v)
			h_dist[v] = INFINITY;
	}

	real_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(real_t) * h_mesh.n_vertices);
	hipMalloc(&d_dist[1], sizeof(real_t) * h_mesh.n_vertices);

	index_t * d_sorted = nullptr;
	if(!coalescence)
	{
		hipMalloc(&d_sorted, sizeof(index_t) * h_mesh.n_vertices);
	}

	real_t * d_error;
	hipMalloc(&d_error, sizeof(real_t) * h_mesh.n_vertices);


	index_t * h_clusters = coalescence && ptp_out.clusters ? new index_t[h_mesh.n_vertices]
															: ptp_out.clusters;

	index_t * d_clusters[2] = {};

	if(h_clusters)
	{
		hipMalloc(&d_clusters[0], sizeof(index_t) * h_mesh.n_vertices);
		hipMalloc(&d_clusters[1], sizeof(index_t) * h_mesh.n_vertices);
	}

	const index_t & d = run_ptp_gpu(d_mesh, sources, h_mesh.n_vertices,
									h_dist, d_dist,
									{toplesets.limits, coalescence ? inv : toplesets.index},
									d_error,
									h_clusters, d_clusters,
									d_sorted);

	hipMemcpy(h_dist, d_dist[d], sizeof(real_t) * h_mesh.n_vertices, hipMemcpyDeviceToHost);
	if(coalescence)
	{
		#pragma omp parallel for
		for(index_t i = 0; i < toplesets.limits.back(); ++i)
			ptp_out.dist[toplesets.index[i]] = h_dist[i];

		delete [] h_dist;
	}

	if(h_clusters)
	{
		hipMemcpy(h_clusters, d_clusters[d], sizeof(index_t) * h_mesh.n_vertices, hipMemcpyDeviceToHost);

		if(coalescence)
		{
			#pragma omp parallel for
			for(index_t i = 0; i < h_mesh.n_vertices; ++i)
				ptp_out.clusters[toplesets.index[i]] = h_clusters[i];

			delete [] h_clusters;
		}

		hipFree(d_clusters[0]);
		hipFree(d_clusters[1]);
	}

	hipFree(d_error);
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	cuda_free_CHE(dd_mesh, d_mesh);

	if(coalescence)
	{
		delete [] h_mesh.GT;
		delete [] h_mesh.VT;
		delete [] h_mesh.EVT;
	}
	else
	{
		hipFree(d_sorted);
	}

	delete [] inv;

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float time;
	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return time / 1000;
}

index_t run_ptp_gpu(const CHE * d_mesh, const std::vector<index_t> & sources, const index_t & n_vertices,
					real_t * h_dist, real_t ** d_dist, const toplesets_t & inv, real_t * d_error,
					index_t * h_clusters, index_t ** d_clusters, index_t * d_sorted)
{
	for(index_t i = 0; i < sources.size(); ++i)
	{
		const index_t & s = sources[i];
		const index_t & v = d_sorted ? s: inv.index[s];

		h_dist[v] = 0;

		if(h_clusters)
			h_clusters[v] = i;
	}

	hipMemcpy(d_dist[0], h_dist, sizeof(real_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_dist[1], h_dist, sizeof(real_t) * n_vertices, hipMemcpyHostToDevice);

	if(d_sorted)
	{
		hipMemcpy(d_sorted, inv.index, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);
	}


	if(h_clusters)
	{
		hipMemcpy(d_clusters[0], h_clusters, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);
		hipMemcpy(d_clusters[1], h_clusters, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);
	}

	index_t d = 0;
	index_t start, end, n_cond;
	index_t i = 1, j = 2;

	// maximum number of iterations
	index_t iter = 0;
	index_t max_iter = inv.limits.size() << 1;

	while(i < j && iter++ < max_iter)
	{
		if(i < (j >> 1)) i = (j >> 1); // K/2 limit band size

		start = inv.limits[i];
		end = inv.limits[j];
		n_cond = inv.limits[i + 1] - start;

		h_clusters ? relax_ptp<<< NB(end - start), NT >>>(d_mesh, d_dist[!d], d_dist[d], d_clusters[!d], d_clusters[d], start, end, d_sorted)
					: relax_ptp<<< NB(end - start), NT >>>(d_mesh, d_dist[!d], d_dist[d], nullptr, nullptr, start, end, d_sorted);

		hipDeviceSynchronize();

		relative_error<<< NB(n_cond), NT >>>(d_error, d_dist[!d], d_dist[d], start, start + n_cond);
		hipDeviceSynchronize();

		if(n_cond == thrust::count_if(thrust::device, d_error + start, d_error + start + n_cond, is_ok()))
			++i;

		if(j < inv.limits.size() - 1) ++j;

		d = !d;
	}

	return d;
}

real_t farthest_point_sampling_ptp_gpu(che * mesh, std::vector<index_t> & samples, double & time_fps, size_t n, real_t radio)
{
	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// BEGIN FPS PTP

	CHE * h_mesh = new CHE(mesh);
	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	real_t * h_dist = new real_t[h_mesh->n_vertices];
	#pragma omp parallel for
	for(index_t v = 0; v < h_mesh->n_vertices; ++v)
		h_dist[v] = INFINITY;


	real_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(real_t) * h_mesh->n_vertices);
	hipMalloc(&d_dist[1], sizeof(real_t) * h_mesh->n_vertices);

	real_t * d_error;
	hipMalloc(&d_error, sizeof(real_t) * h_mesh->n_vertices);

	index_t * d_sorted;
	hipMalloc(&d_sorted, sizeof(index_t) * h_mesh->n_vertices);

	std::vector<index_t> limits;
	index_t * toplesets = new index_t[h_mesh->n_vertices];
	index_t * sorted_index = new index_t[h_mesh->n_vertices];

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	if(n >= h_mesh->n_vertices) n = h_mesh->n_vertices >> 1;

	n -= samples.size();
	samples.reserve(n);

	index_t d;
	int f;
	real_t max_dist = INFINITY;
	while(n-- && max_dist > radio)
	{
		limits.clear();
		mesh->compute_toplesets(toplesets, sorted_index, limits, samples);

		d = run_ptp_gpu(d_mesh, samples, h_mesh->n_vertices, h_dist, d_dist, {limits, sorted_index}, d_error, nullptr, nullptr, d_sorted);

		// 1 indexing
		#ifdef GPROSHAN_FLOAT
			hipblasIsamax(handle, mesh->n_vertices, d_dist[d], 1, &f);
		#else
			hipblasIdamax(handle, mesh->n_vertices, d_dist[d], 1, &f);
		#endif

		if(radio > 0 || !n)
			hipMemcpy(&max_dist, d_dist[d] + f - 1, sizeof(real_t), hipMemcpyDeviceToHost);

		samples.push_back(f - 1);
	}

	hipblasDestroy(handle);

	delete [] h_dist;
	delete [] toplesets;
	delete [] sorted_index;

	hipFree(d_error);
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	hipFree(d_sorted);
	cuda_free_CHE(dd_mesh, d_mesh);

	// END FPS PTP
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	time_fps = time / 1000;

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return max_dist;
}

__global__
void relax_ptp(const CHE * mesh, real_t * new_dist, real_t * old_dist, index_t * new_clusters, index_t * old_clusters, const index_t start, const index_t end, const index_t * sorted)
{
	index_t v = blockDim.x * blockIdx.x + threadIdx.x + start;

	if(v < end)
		relax_ptp(mesh, new_dist, old_dist, new_clusters, old_clusters, sorted ? sorted[v] : v);
}

__global__
void relative_error(real_t * error, const real_t * new_dist, const real_t * old_dist, const index_t start, const index_t end, const index_t * sorted)
{
	index_t i = blockDim.x * blockIdx.x + threadIdx.x + start;

	if(i < end)
	{
		index_t v = sorted ? sorted[i] : i;

		#ifdef GPROSHAN_FLOAT
			error[i] = fabsf(new_dist[v] - old_dist[v]) / old_dist[v];
		#else
			error[i] = fabs(new_dist[v] - old_dist[v]) / old_dist[v];
		#endif
	}
}

__host__ __device__
bool is_ok::operator()(const real_t & val) const
{
	return val < PTP_TOL;
}


} // namespace gproshan

