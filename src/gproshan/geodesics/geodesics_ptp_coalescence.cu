#include <gproshan/geodesics/geodesics_ptp_coalescence.cuh>

#include <gproshan/mesh/che_off.h>

#include <cstdio>
#include <fstream>
#include <cassert>
#include <hipblas.h>

#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>


// geometry processing and shape analysis framework
namespace gproshan {


double parallel_toplesets_propagation_coalescence_gpu(const ptp_out_t & ptp_out, const che * mesh, const std::vector<index_t> & sources, const toplesets_t & toplesets, const bool & set_inf)
{
	index_t * inv = nullptr;
	che * coalescence_mesh = ptp_coalescence(inv, mesh, toplesets);

	// ------------------------------------------------------

	hipDeviceReset();

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// BEGIN PTP

	CHE * h_mesh = new CHE(coalescence_mesh);
	CHE * dd_mesh, * d_mesh;
	cuda_create_CHE(h_mesh, dd_mesh, d_mesh);

	real_t * h_dist = new real_t[h_mesh->n_vertices];

	if(set_inf)
	{
		#pragma omp parallel for
		for(index_t v = 0; v < h_mesh->n_vertices; ++v)
			h_dist[v] = INFINITY;
	}

	real_t * d_dist[2];
	hipMalloc(&d_dist[0], sizeof(real_t) * h_mesh->n_vertices);
	hipMalloc(&d_dist[1], sizeof(real_t) * h_mesh->n_vertices);

	real_t * d_error;
	hipMalloc(&d_error, sizeof(real_t) * h_mesh->n_vertices);

	index_t d;
	if(ptp_out.clusters)
	{
		index_t * h_clusters = new index_t[h_mesh->n_vertices];
		index_t * d_clusters[2] = {nullptr, nullptr};

		hipMalloc(&d_clusters[0], sizeof(index_t) * h_mesh->n_vertices);
		hipMalloc(&d_clusters[1], sizeof(index_t) * h_mesh->n_vertices);

		d = run_ptp_coalescence_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, sources, {toplesets.limits, inv}, d_error, h_clusters, d_clusters);
		hipMemcpy(h_clusters, d_clusters[d], sizeof(index_t) * h_mesh->n_vertices, hipMemcpyDeviceToHost);

		#pragma omp parallel for
		for(index_t i = 0; i < h_mesh->n_vertices; ++i)
			ptp_out.clusters[toplesets.index[i]] = h_clusters[i];

		hipFree(d_clusters[0]);
		hipFree(d_clusters[1]);

		delete [] h_clusters;
	}
	else d = run_ptp_coalescence_gpu(d_mesh, h_mesh->n_vertices, h_dist, d_dist, sources, {toplesets.limits, inv}, d_error);

	hipMemcpy(h_dist, d_dist[d], sizeof(real_t) * h_mesh->n_vertices, hipMemcpyDeviceToHost);

	hipFree(d_error);
	hipFree(d_dist[0]);
	hipFree(d_dist[1]);
	cuda_free_CHE(dd_mesh, d_mesh);

	delete coalescence_mesh;
	delete [] inv;

	#pragma omp parallel for
	for(index_t i = 0; i < toplesets.limits.back(); ++i)
		ptp_out.dist[toplesets.index[i]] = h_dist[i];

	delete [] h_dist;

	// END PTP

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return time / 1000;
}

index_t run_ptp_coalescence_gpu(CHE * d_mesh, const index_t & n_vertices, real_t * h_dist, real_t ** d_dist, const std::vector<index_t> & sources, const toplesets_t & inv, real_t * d_error, index_t * h_clusters, index_t ** d_clusters)
{
	for(index_t i = 0; i < sources.size(); ++i)
		h_dist[inv.index[sources[i]]] = 0;

	hipMemcpy(d_dist[0], h_dist, sizeof(real_t) * n_vertices, hipMemcpyHostToDevice);
	hipMemcpy(d_dist[1], h_dist, sizeof(real_t) * n_vertices, hipMemcpyHostToDevice);

	if(h_clusters)
	{
		assert(d_clusters[0]);

		for(index_t i = 0; i < sources.size(); ++i)
			h_clusters[inv.index[sources[i]]] = i + 1;

		hipMemcpy(d_clusters[0], h_clusters, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);
		hipMemcpy(d_clusters[1], h_clusters, sizeof(index_t) * n_vertices, hipMemcpyHostToDevice);
	}

	index_t d = 0;
	index_t start, end, n_cond;
	index_t i = 1, j = 2;

	// maximum number of iterations
	index_t iter = 0;
	index_t max_iter = inv.limits.size() << 1;

	while(i < j && iter++ < max_iter)
	{
		if(i < (j >> 1)) i = (j >> 1); // K/2 limit band size

		start = inv.limits[i];
		end = inv.limits[j];
		n_cond = inv.limits[i + 1] - start;

		h_clusters ? relax_ptp <<< NB(end - start), NT >>> (d_mesh, d_dist[!d], d_dist[d], d_clusters[!d], d_clusters[d], start, end)
					: relax_ptp <<< NB(end - start), NT >>> (d_mesh, d_dist[!d], d_dist[d], nullptr, nullptr, start, end);

		hipDeviceSynchronize();

		relative_error <<< NB(n_cond), NT >>>(d_error, d_dist[!d], d_dist[d], start, start + n_cond);
		hipDeviceSynchronize();

		if(n_cond == thrust::count_if(thrust::device, d_error + start, d_error + start + n_cond, is_ok()))
			++i;

		if(j < inv.limits.size() - 1) ++j;

		d = !d;
	}

	return d;
}


} // namespace gproshan

