#include "hip/hip_runtime.h"
#include <gproshan/mesh/che.cuh>
#include <gproshan/raytracing/utils.h>
#include <gproshan/raytracing/optix_params.h>

#include <optix_device.h>
#include <hip/hip_runtime.h>


// geometry processing and shape analysis framework
namespace gproshan::rt {


extern "C" __constant__ launch_params optix_params;

static __forceinline__ __device__
void * unpack_pointer(uint32_t i0, uint32_t i1)
{
	return (void *) (uint64_t(i0) << 32 | i1);
}

static __forceinline__ __device__
void pack_pointer(void * ptr, uint32_t & i0, uint32_t & i1)
{
	const uint64_t uptr = uint64_t(ptr);
	i0 = uptr >> 32;
	i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__
T * ray_data()
{
	return (T *) unpack_pointer(optixGetPayload_0(), optixGetPayload_1());
}


extern "C" __global__ void __closesthit__shadow() {}

extern "C" __global__ void __closesthit__radiance()
{
	const CHE & mesh = **(const CHE **) optixGetSbtDataPointer();

	const int primID = optixGetPrimitiveIndex();
	float2 bar = optixGetTriangleBarycentrics();

	OptixTraversableHandle gas = optixGetGASTraversableHandle();
	const index_t sbtID = optixGetSbtGASIndex();
	const float time = optixGetRayTime();

	vertex data[3];
	optixGetTriangleVertexData(gas, primID, sbtID, time, (float3 *) data);

	const vertex & A = data[0];
	const vertex & B = data[1];
	const vertex & C = data[2];

	eval_hit hit(mesh, primID, bar.x, bar.y, optix_params.sc);
	hit.normal = optix_params.flat ? normalize(cross(B - A, C - A)) : hit.normal;
	hit.position = (1.f - hit.u - hit.v) * A + hit.u * B + hit.v * C;

	vec3 * trace = ray_data<vec3>();
	vec3 & color		= trace[0];
	vec3 & position		= trace[1];
	vec3 & scattered	= trace[2];	// in ray_dir / out scattered
	vec3 & attenuation	= trace[3];

	color = eval_li(hit, optix_params.ambient, optix_params.lights, optix_params.n_lights, optix_params.cam_pos,
					[&](const vec3 & position, const vec3 & wi, const float & light_dist) -> bool
					{
						uint32_t occluded = 1;
						optixTrace( optix_params.traversable,
									* (float3 *) &position,
									* (float3 *) &wi,
									1e-3f,					// tmin
									light_dist - 1e-3f,		// tmax
									0.0f,					// rayTime
									OptixVisibilityMask(255),
										OPTIX_RAY_FLAG_DISABLE_ANYHIT
										| OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT
										| OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
										1,	// SBT offset
										2,	// SBT stride
										1,	// missSBTIndex
										occluded);

							return occluded != 0;
						});

	color *= attenuation;
	position = hit.position;

	if(!hit.scatter_mat(scattered, scattered))
		attenuation = 0;
}


extern "C" __global__ void __anyhit__radiance() {}

extern "C" __global__ void __anyhit__shadow() {}


extern "C" __global__ void __miss__radiance()
{
	vec4 & pixel_color = *ray_data<vec4>();
	pixel_color = {0, 0, 0, 0};
}

extern "C" __global__ void __miss__shadow()
{
	optixSetPayload_0(0);
}


extern "C" __global__ void __raygen__render_frame()
{
	const int ix = optixGetLaunchIndex().x;
	const int iy = optixGetLaunchIndex().y;

	vec3 trace[4];
	vec3 & color		= trace[0];
	vec3 & position		= trace[1];
	vec3 & ray_dir		= trace[2];
	vec3 & attenuation	= trace[3] = 1;

	position = optix_params.cam_pos;
	ray_dir = ray_view_dir(	{ix + optix_params.viewport_x, iy + optix_params.viewport_y},
							{optix_params.window_width, optix_params.window_height},
							optix_params.inv_proj_view,
							optix_params.cam_pos
							);

	uint32_t u0, u1;
	pack_pointer(trace, u0, u1);

	int depth = 3;
	vec3 color_acc = 0;

	while(--depth)
	{
		optixTrace(	optix_params.traversable,
					* (float3 *) &position,
					* (float3 *) &ray_dir,
					1e-5f,	// tmin
					1e20f,	// tmax
					0.0f,	// rayTime
					OptixVisibilityMask(255),
					OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
					0,	// SBT offset
					2,	// SBT stride
					0,	// missSBTIndex
					u0, u1);

		color_acc += color;
	}

	vec4 & pixel_color = optix_params.color_buffer[ix + iy * optixGetLaunchDimensions().x];
	pixel_color = (pixel_color * optix_params.n_samples + (color_acc, 1)) / (optix_params.n_samples + 1);
}


} // namespace gproshan

