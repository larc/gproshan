#include "hip/hip_runtime.h"
#include <gproshan/mesh/che.cuh>
#include <gproshan/raytracing/rt_utils.h>
#include <gproshan/raytracing/rt_optix_params.h>


#include <optix_device.h>
#include <hip/hip_runtime.h>


// geometry processing and shape analysis framework
namespace gproshan::rt {


extern "C" __constant__ launch_params optix_params;

static __forceinline__ __device__
void * unpack_pointer(uint32_t i0, uint32_t i1)
{
	return (void *) (uint64_t(i0) << 32 | i1);
}

static __forceinline__ __device__
void pack_pointer(void * ptr, uint32_t & i0, uint32_t & i1)
{
	const uint64_t uptr = uint64_t(ptr);
	i0 = uptr >> 32;
	i1 = uptr & 0x00000000ffffffff;
}

template<typename T>
static __forceinline__ __device__
T * ray_data()
{
	return (T *) unpack_pointer(optixGetPayload_0(), optixGetPayload_1());
}


extern "C" __global__ void __closesthit__shadow() {}

extern "C" __global__ void __closesthit__radiance()
{
	const CHE & mesh = **(const CHE **) optixGetSbtDataPointer();

	const unsigned int primID = optixGetPrimitiveIndex();

	const int he = primID * che::mtrig;
	const float u = optixGetTriangleBarycentrics().x;
	const float v = optixGetTriangleBarycentrics().y;

	const int a = mesh.VT[he];
	const int b = mesh.VT[he + 1];
	const int c = mesh.VT[he + 2];

	OptixTraversableHandle gas = optixGetGASTraversableHandle();
	const unsigned int sbtID = optixGetSbtGASIndex();
	const float time = optixGetRayTime();

	vertex data[3];
	optixGetTriangleVertexData(gas, primID, sbtID, time, (float3 *) data);

	const vertex & A = data[0];
	const vertex & B = data[1];
	const vertex & C = data[2];

	const vertex normal = optix_params.flat ? normalize((B - A) * (C - A))
											: (1.f - u - v) * mesh.VN[a] + u * mesh.VN[b] + v * mesh.VN[c];

	const vertex ca = {float(mesh.VC[a].r), float(mesh.VC[a].g), float(mesh.VC[a].b)};
	const vertex cb = {float(mesh.VC[b].r), float(mesh.VC[b].g), float(mesh.VC[b].b)};
	const vertex cc = {float(mesh.VC[c].r), float(mesh.VC[c].g), float(mesh.VC[c].b)};

	const vertex * lights = optix_params.lights;
	const vertex color = ((1.f - u - v) * ca + u * cb + v * cc) / 255;
	const vertex position = (1.f - u - v) * A + u * B + v * C;

	vertex & L = *ray_data<vertex>();

	L = {0, 0, 0};
	for(int i = 0; i < optix_params.n_lights; ++i)
	{
		vertex wi = lights[i] - position;
		float light_dist = length(wi);
		wi /= light_dist;
		float dot_wi_normal = (wi, normal);

		unsigned int occluded = 1;
		optixTrace( optix_params.traversable,
					* (float3 *) &position,
					* (float3 *) &wi,
					1e-3f,					// tmin
					light_dist - 1e-3f,		// tmax
					0.0f,					// rayTime
					OptixVisibilityMask(255),
					OPTIX_RAY_FLAG_DISABLE_ANYHIT
					| OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT
					| OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
					1,	// SBT offset
					2,	// SBT stride
					1,	// missSBTIndex
					occluded);

		L += (dot_wi_normal < 0 ? -dot_wi_normal : dot_wi_normal) * (occluded ? 0.4f : 1.0f) * color;
	}

	L /= optix_params.n_lights;
}


extern "C" __global__ void __anyhit__radiance() {}

extern "C" __global__ void __anyhit__shadow() {}


extern "C" __global__ void __miss__radiance()
{
	vec3 & pixel_color = *ray_data<vertex>();
	pixel_color = {0, 0, 0};
}

extern "C" __global__ void __miss__shadow()
{
	optixSetPayload_0(0);
}


extern "C" __global__ void __raygen__render_frame()
{
	const int ix = optixGetLaunchIndex().x;
	const int iy = optixGetLaunchIndex().y;

	const vec3 ray_dir = ray_view_dir(	{ix + optix_params.viewport_x, iy + optix_params.viewport_y},
										{optix_params.window_width, optix_params.window_height},
										optix_params.inv_proj_view,
										optix_params.cam_pos
										);

	vec4 & pixel_color = optix_params.color_buffer[ix + iy * optixGetLaunchDimensions().x];

	uint32_t u0, u1;
	pack_pointer(&pixel_color, u0, u1);

	optixTrace(	optix_params.traversable,
				* (float3 *) &optix_params.cam_pos,
				* (float3 *) &ray_dir,
				0.f,	// tmin
				1e20f,	// tmax
				0.0f,	// rayTime
				OptixVisibilityMask(255),
				OPTIX_RAY_FLAG_DISABLE_ANYHIT, //OPTIX_RAY_FLAG_NONE,
				0,	// SBT offset
				2,	// SBT stride
				0,	// missSBTIndex
				u0, u1);

	pixel_color[3] = 1;
}


} // namespace gproshan

