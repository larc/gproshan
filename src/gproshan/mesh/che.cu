#include <gproshan/mesh/che.cuh>


// geometry processing and shape analysis framework
namespace gproshan {


void cuda_create_CHE(const che * h_che, che *& dd_che, che *& d_che, const bool normal, const bool color)
{
	dd_che = new che;
	dd_che->n_vertices = h_che->n_vertices;
	dd_che->n_trigs = h_che->n_trigs;
	dd_che->n_half_edges = h_che->n_half_edges;

	hipMalloc(&dd_che->GT, sizeof(vertex) * h_che->n_vertices);
	hipMemcpy(dd_che->GT, h_che->GT, sizeof(vertex) * h_che->n_vertices, hipMemcpyHostToDevice);

	if(normal)
	{
		hipMalloc(&dd_che->VN, sizeof(vertex) * h_che->n_vertices);
		hipMemcpy(dd_che->VN, h_che->VN, sizeof(vertex) * h_che->n_vertices, hipMemcpyHostToDevice);
	}

	if(color)
	{
		hipMalloc(&dd_che->VC, sizeof(che::rgb_t) * h_che->n_vertices);
		hipMemcpy(dd_che->VC, h_che->VC, sizeof(che::rgb_t) * h_che->n_vertices, hipMemcpyHostToDevice);

		hipMalloc(&dd_che->VHC, sizeof(real_t) * h_che->n_vertices);
		hipMemcpy(dd_che->VHC, h_che->VHC, sizeof(real_t) * h_che->n_vertices, hipMemcpyHostToDevice);
	}

	hipMalloc(&dd_che->VT, sizeof(index_t) * h_che->n_half_edges);
	hipMemcpy(dd_che->VT, h_che->VT, sizeof(index_t) * h_che->n_half_edges, hipMemcpyHostToDevice);

	hipMalloc(&dd_che->OT, sizeof(index_t) * h_che->n_half_edges);
	hipMemcpy(dd_che->OT, h_che->OT, sizeof(index_t) * h_che->n_half_edges, hipMemcpyHostToDevice);

	hipMalloc(&dd_che->EVT, sizeof(index_t) * h_che->n_vertices);
	hipMemcpy(dd_che->EVT, h_che->EVT, sizeof(index_t) * h_che->n_vertices, hipMemcpyHostToDevice);

	hipMalloc(&d_che, sizeof(che));
	hipMemcpy(d_che, dd_che, sizeof(che), hipMemcpyHostToDevice);
}

void cuda_free_CHE(che *& dd_che, che *& d_che)
{
	hipFree(dd_che->GT);
	hipFree(dd_che->EVT);
	hipFree(dd_che->VT);
	hipFree(dd_che->OT);
	hipFree(dd_che->VN);
	hipFree(dd_che->VC);
	hipFree(dd_che->VHC);

	//delete dd_che;
	hipFree(d_che);
}


} // namespace gproshan

